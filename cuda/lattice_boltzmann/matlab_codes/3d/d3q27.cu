
#include <hip/hip_runtime.h>
/**
* @file streamingd3q27.cu
*
* CUDA code to perform 3D LBM simulation on a GPU.
*
* Copyright Yuxuan Liu 2019
*/
        
/** Work out which piece of the global array this thread should operate on */
__device__ size_t calculateGlobalIndex() {
    // Which block are we?
    size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
    // Which thread are we within the block?
    size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
    // How big is each block?
    size_t const threadsPerBlock = blockDim.x*blockDim.y;
    // Which thread are we overall?
    return localThreadIdx + globalBlockIndex*threadsPerBlock;

}



/** Kernel constants */
__constant__ int tileX;
__constant__ int tileY;
__constant__ int tileZ;
__constant__ int nElem;
__constant__ int nDir;
__constant__ double wi[27];
__constant__ int dirX[27];
__constant__ int dirY[27];
__constant__ int dirZ[27];
__constant__ int bi[27];

__constant__ double cs[4];// csSqrInv, csSqr, csInv, cs
__constant__ double tau[3];// tau, 1/tau, 1-1/tau

// Periodic fetching
__device__ __forceinline__
int2 fetch2D(int ind) {

    int2 ij;
    ij.x = (ind%nElem)%tileX;
    ij.y = (ind%nElem)/tileX;
    return ij;

};

__device__ __forceinline__
int3 fetch3D(int ind) {

    int3 ijk;
    ijk.x = (ind%(tileX*tileY))%tileX;
    ijk.y = (ind%(tileX*tileY))/tileX;
    ijk.z = (ind%nElem)/(tileX*tileY);
    return ijk;

};

// Periodic fetching
__device__ __forceinline__
int fetchInd2(int2 ij) {

    return (ij.x + tileX*ij.y + nElem)%nElem;

};
__device__ __forceinline__
int fetchInd(int3 ijk) {

    return (ijk.x + tileX*(ijk.y + tileY*ijk.z) + nElem)%nElem;

};

// Periodic fetching
__device__ __forceinline__
int fetchStencil2(int2 ij, int dirx, int diry) {

    return (ij.x+dirx + tileX*(ij.y+diry) + nElem)%nElem;

};
__device__ __forceinline__
int fetchStencil(int3 ijk, int dirx, int diry, int dirz) {

    return (ijk.x+dirx + tileX*(ijk.y+diry  + tileY*(ijk.z + dirz)) + nElem)%nElem;

};

/** Streaming kernal */
__device__ void streaming2(int * out, const int2& uv)
{
    for(int i = 0; i < nDir; i++)
    {
        int2 dir;
        dir.x = uv.x - dirX[i];
        dir.y = uv.y - dirY[i];
        out[i] = fetchInd2(dir);
    }
}
__device__ void streaming(int * out, const int3& uvw)
{
    for(int i = 0; i < nDir; i++)
    {
        int3 dir;
        dir.x = uvw.x - dirX[i];
        dir.y = uvw.y - dirY[i];
        dir.z = uvw.z - dirZ[i];
        out[i] = fetchInd(dir);
    }
}

/** Main entry point.
 * Works out where the current thread should read/write to global memory
 * and calls doIterations to do the actual work.
 */
__global__ void processStreaming(
                      double * out,
                      const double * in,
                      const double * r,
                      const double * u,
                      const double * v,
                      const double * w,
                      const bool * mask ) {
    // Work out which thread we are
    size_t const globalThreadIdx = calculateGlobalIndex();
    int3 ijk = fetch3D(globalThreadIdx);

    // If we're off the end, return now
    if (globalThreadIdx >= nElem) {
        return;
    }

    // Get our X and Y coords
    int neighbours[27];
    streaming(neighbours, ijk);

    // Processing boundary nodes
    if(mask[globalThreadIdx] == true) {
        //double req = r[globalThreadIdx];
        //double ueq = u[globalThreadIdx];
        //double veq = v[globalThreadIdx];
        //double weq = w[globalThreadIdx];

        // no-slip (Bounce_back)
        //for(int i = 0; i < nDir; i++)
        //{
        //    double cidotw = dirX[i]*ueq + dirY[i]*veq + dirZ[i]*weq;

         // rhow ignored for incompressible flow
         //   out[globalThreadIdx + bi[i]*nElem] =
         //       in[neighbours[i] + i*nElem] - 2.0*wi[i]*req*(3.0*cidotw);
        //}

        return;
    }

    // Processing internal nodes
    for(int i = 0; i < nDir; i++)
    {
        out[globalThreadIdx + i*nElem] = in[neighbours[i] + i*nElem];
    }

}

/** Main entry point.
 * Works out where the current thread should read/write to global memory
 * and calls doIterations to do the actual work.
 */
__device__ double compute_equilibrium(
                          const double& r,
                          const double& u,
                          const double& v,
                          const double& w,
                          const int& dim) {

        // calculate dot product
        double cidotu = dirX[dim]*u + dirY[dim]*v + dirZ[dim]*w;

        // calculate equilibrium
        return wi[dim]*r*(1.0 + 3.0*cidotu+4.5*cidotu*cidotu-1.5*(u*u+v*v+w*w));

}

__device__ __forceinline__ double collision(
                          const double& f,
                          const double& feq,
                          const double& a,
                          const double& b) {

        return (f - a*b*(f - feq));

}

__device__ double compute_g(
                          const double * f,
                          const double * feq,
                          const double& a,const double& b) {

        double result = 0.0;
        for(int i = 0;i < nDir; i++)
        {
            double c = collision(f[i],feq[i],a,b);
            if(c < 0.0) {c = 0.00000001;}
            result += c*log(c/wi[i]) - f[i]*log(f[i]/wi[i]);
        }
        return result;
}

__device__ double compute_gradg(
                          const double * f,
                          const double * feq,
                          const double& a,const double& b)
{

        double result = 0.0;
        for(int i = 0;i < nDir; i++)
        {
            double c = collision(f[i],feq[i],a,b);
            if(c < 0.0) {c = 0.00000001;}
            result += -b*(f[i] - feq[i])*(log(c/wi[i]) + 1.0);
        }
        return result;
}

__device__ __forceinline__ void swap(double& a, double& b)
{

            double tmp = a;
            a = b;
            b = tmp;
}

__constant__ double stableDeviation;
__constant__ double alphaMin;
__constant__ bool enableEntropyConstraint;
__device__ double constrain_entropy(
                          const double * f,
                          const double * feq,const double& b) {
        // calculate deviation
        double amin=alphaMin, amax=2.0;
        double maxDeviation = 0.0;
        for(int i = 0;i < nDir; i++)
        {
            double deviation = abs(f[i]/feq[i]-1.0);
            if(deviation > maxDeviation)
                maxDeviation = deviation;
        }

        // if deviation is too large
        //double stableDeviation = 0.2;
        if(maxDeviation < stableDeviation) return amax;

        // compute G value
        double Gmin = compute_g(f,feq,amin,b);
        double Gmax = compute_g(f,feq,amax,b);
        double gradGmin = compute_gradg(f,feq,amin,b);
        double gradGmax = compute_gradg(f,feq,amax,b);
        if(Gmin*Gmax > 0) return amax;
        if(Gmin > 0) swap(amin, amax);

        double a = 0.5*(amin + amax);
        double da = abs(amax - amin);
        double a_o = a;
        //double da_o = da;
        double G = compute_g(f,feq,a,b);
        double gradG = compute_gradg(f,feq,a,b);

        int maxIter = 20;
        double tolerance = 0.0001;
        for(int it = 0; it < maxIter; it++)
        {
            if( ( ((a-amax)*gradG-G)*((a-amin)*gradG-G) >= 0 )
            ||  ( abs(a_o*gradG-G-1.0) > 1.0 ) )
            {
                // bisection
                //da_o = da;
                da = 0.5*(amax - amin);
                a = amin-amax;
                if(amin == a) return a;
            }else
            {
                //da_o = da;
                da = G/gradG;
                a_o = a;
                a -= da;
                if(a_o == a) return a;
            }
            if(abs(da) < tolerance) return a;

            G = compute_g(f,feq,a,b);
            gradG = compute_gradg(f,feq,a,b);
            if(G < 0.0) {amin = a;}
            else {amax = a;}
        }

        return amax;

}

__constant__ double constant_Fx[27]; // Fi = wi*c_ia*Fa/cs^2
__constant__ double constant_Fy[27];
__constant__ double constant_Fz[27];

__global__ void processCollision(
                      double * out,
                      const double * in,
                      const double * r,
                      const double * u,
                      const double * v,
                      const double * w,
                      const bool * mask )
{
    // Work out which thread we are
    size_t const globalThreadIdx = calculateGlobalIndex();
    int3 ijk = fetch3D(globalThreadIdx);

    // If we're off the end, return now
    if (globalThreadIdx >= nElem) {
        return;
    }

    // no collision for boundary nodes
    if(mask[globalThreadIdx] == true) {
        return;
    }

    // Read inputs
    double rho_eq = r[globalThreadIdx];
    double u_eq = u[globalThreadIdx];
    double v_eq = v[globalThreadIdx];
    double w_eq = w[globalThreadIdx];

    // Force: Shan&Chen scheme
    for(int i = 0; i < nDir; i++)
    {
        u_eq += (tau[0]-0.5)*constant_Fx[i]*dirX[i]/rho_eq;
        v_eq += (tau[0]-0.5)*constant_Fy[i]*dirY[i]/rho_eq;
        w_eq += (tau[0]-0.5)*constant_Fz[i]*dirZ[i]/rho_eq;
    }

    // Compute collision
    double f[27];
    double feq[27];
    for(int i = 0; i < nDir; i++)
    {
        f[i] = in[globalThreadIdx + i*nElem];
        feq[i] = compute_equilibrium(rho_eq, u_eq, v_eq, w_eq, i);
    }

    // Entropic LBM implementation
    double alpha = 2.0;
    if(enableEntropyConstraint)
    	alpha = constrain_entropy(f, feq, 0.5*tau[1]);

    for(int i = 0; i < nDir; i++)
    {
        // BGK Collider
        f[i] = collision(f[i],feq[i],alpha,0.5*tau[1]);

        out[globalThreadIdx + i*nElem] = f[i];
    }
    //out2[globalThreadIdx] = alpha;

}

/** Main entry point.
 * Works out where the current thread should read/write to global memory
 * and calls doIterations to do the actual work.
 */
__global__ void processCompute(
                      double * r,
                      double * u,
                      double * v,
                      double * w,
                      const double * fin,
                      const bool * mask ) {
    // Work out which thread we are
    size_t const globalThreadIdx = calculateGlobalIndex();
    int3 ijk = fetch3D(globalThreadIdx);

    // If we're off the end, return now
    if (globalThreadIdx >= nElem) {
        return;
    }

    // no collision for boundary nodes
    if(mask[globalThreadIdx] == true) {
        return;
    }

    // compute r, u, v
    double _rho=0, _u=0, _v=0, _w=0;
    for(int i = 0; i < nDir; i++)
    {
        double f = fin[globalThreadIdx + i*nElem];
        _rho += f;
        _u += dirX[i]*f;
        _v += dirY[i]*f;
        _w += dirZ[i]*f;

        // Force contribution
        _u += 0.5*constant_Fx[i]*dirX[i];
        _v += 0.5*constant_Fy[i]*dirY[i];
        _w += 0.5*constant_Fz[i]*dirZ[i];
    }

    _u /= _rho;
    _v /= _rho;
    _w /= _rho;

    // Output
    r[globalThreadIdx] = _rho;
    u[globalThreadIdx] = _u;
    v[globalThreadIdx] = _v;
    w[globalThreadIdx] = _w;

}

__global__ void processPeriodic(
              double * out,
              const double * in,
              const double * a,
              const double * rp,
              const double * r,
              const double * u,
              const double * v, 
              const double * w,
              const int nBElem) {
    // Work out which thread we are
    size_t const globalThreadIdx = calculateGlobalIndex();
    int3 ijk = fetch3D(globalThreadIdx);

    // If we're off the end, return now
    if (globalThreadIdx >= nBElem) {
        return;
    }

    // Read inputs
    double alpha = a[globalThreadIdx];
    double rhop_eq = rp[globalThreadIdx];
    double rho_eq = r[globalThreadIdx];
    double u_eq = u[globalThreadIdx];
    double v_eq = v[globalThreadIdx];
    double w_eq = w[globalThreadIdx];
    double u_eq_s = alpha*u_eq;
    double v_eq_s = alpha*v_eq;
    double w_eq_s = alpha*w_eq;

    // Processing boundary nodes
        // periodic with no pressure drop
        for(int i = 0; i < nDir; i++)
        {
            double f = in[globalThreadIdx + i*nBElem];
            double feq = compute_equilibrium(rho_eq, u_eq, v_eq, w_eq, i);
            double feq_s = compute_equilibrium(rhop_eq, u_eq_s, v_eq_s, w_eq_s, i);

            // rhow ignored for incompressible flow
            out[globalThreadIdx + i*nBElem] = f - feq + feq_s;
        }

}

__global__ void processNoslip(
                      double * out,
                      const double * in,
                      const double * r,
                      const double * u,
                      const double * v,
                      const double * w,
                      const bool * mask ) {
    // Work out which thread we are
    size_t const globalThreadIdx = calculateGlobalIndex();
    int3 ijk = fetch3D(globalThreadIdx);

    // If we're off the end, return now
    if (globalThreadIdx >= nElem) {
        return;
    }
    if(mask[globalThreadIdx] == false) return;

    // Get our X and Y coords
    int neighbours[27];
    streaming(neighbours, ijk);

    // Processing boundary nodes
    double req = r[globalThreadIdx];
    double ueq = u[globalThreadIdx];
    double veq = v[globalThreadIdx];
    double weq = w[globalThreadIdx];

    // no-slip (Bounce_back)
    for(int i = 0; i < nDir; i++)
    {
        double cidotw = dirX[i]*ueq + dirY[i]*veq + dirZ[i]*weq;

      //rhow ignored for incompressible flow
        out[globalThreadIdx + bi[i]*nElem] =
                in[neighbours[i] + i*nElem] - 2.0*wi[i]*req*(3.0*cidotw);
    }

}

__device__ double one_sided_diff(double a1, double a2, double a3, double dir)
{
    return dir*(0.5*(-3.0*a1+4.0*a2-a3));
}

__device__ double central_diff(double ap, double am)
{
    return 0.5*(ap-am);
}

__global__ void processRoutlet(
              double * out,
              const double * in,
              const double * r,
              const double * u,
              const double * v,
              const double * w,
              const bool * mask) {
    // Work out which thread we are
    size_t const globalThreadIdx = calculateGlobalIndex();
    int3 ijk = fetch3D(globalThreadIdx);

    // If we're off the end, return now
    if (globalThreadIdx >= nElem) return;
    if(mask[globalThreadIdx] == false) return;

    // Read stencil
    int x1 = fetchStencil(ijk,-1, 0, 0);
    double r1 = r[x1];
    double u1 = u[x1];
    double v1 = v[x1];
    double w1 = w[x1];

    // Processing boundary nodes
    // periodic with no pressure drop
    for(int i = 0; i < nDir; i++)
    {
        //double feq = compute_equilibrium(r1,u1,v1, i);

        // replace f by feq
        out[globalThreadIdx + i*nElem] = in[x1 + i*nElem];
    }
    
}